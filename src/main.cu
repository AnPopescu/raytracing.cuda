#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <iostream>

#include "scene.h"

#define RESOLUTION 1
#define SAMPLES 5000

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)


void check_cuda(hipError_t result, 
                char const *const func, 
                const char *const file, 
                int const line) {
    if(result){
        std::cerr << "CUDA error = "<< static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}


__device__ vec3 shade(const Ray& r, 
                      Hitable **world, 
                      int depth,
                      hiprandState *state) {
    HitRecord rec;
    if((*world)->hit(r, 0.001, MAXFLOAT, rec)){
        Ray scattered;
        vec3 attenuation;
        vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
        if(depth < 15 && rec.mat_ptr->scatter(r, rec, attenuation, scattered, state)){
            return emitted + attenuation * shade(scattered, world, depth + 1, state);
        }else{
            return emitted;
        }
    }else{
        return vec3(0, 0, 0);
    }
}


__device__ vec3 shade_nolight(const Ray& r, 
                              Hitable **world, 
                              int depth,
                              hiprandState *state) {
    HitRecord rec;
    if((*world)->hit(r, 0.001, MAXFLOAT, rec)){
        Ray scattered;
        vec3 attenuation;
        if(depth < 15 && rec.mat_ptr->scatter(r, rec, attenuation, scattered, state)){
            return attenuation * shade_nolight(scattered, world, depth + 1, state);
        }
        else {
            return vec3(0, 0, 0);
        }
    }else{
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5 * (unit_direction.y() + 1.0);
        return (1.0-t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
}


__global__ void build_scene(Hitable** obj_list, 
                            Hitable** world,
                            Camera** camera,
                            hiprandState *state,
                            int nx,
                            int ny,
                            int cnt){
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        
        // random_scene(obj_list, world, state);
        // simple_light_scene(obj_list, world);
        // cornell_box_scene(obj_list, world);
        cornell_smoke_scene(obj_list, world, state);
        // bvh_scene(obj_list, world, state);
        // final_scene(obj_list, world, state);

        vec3 lookfrom(278, 278, -800);
        vec3 lookat(278, 278, 0);
        float dist_to_focus = 10.0;
        float aperture = 0.0;
        float vfov = 40.0;

        // vec3 lookfrom(13, 2, 3);
        // vec3 lookat(0, 0, 0);
        // float dist_to_focus = 10.0;
        // float aperture = 0.0;
        // float vfov = 20.0;

        *camera = new MotionCamera(lookfrom, 
                                lookat, 
                                vec3(0, 1, 0), 
                                vfov, 
                                float(nx) / float(ny), 
                                aperture, 
                                dist_to_focus,
                                0.0,
                                1.0);
    }
}


__global__ void random_init(int nx, 
                            int ny, 
                            hiprandState *state) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if((x >= nx) || (y >= ny)) return;
    int pixel_index = y * nx + x;
    hiprand_init(0, pixel_index, 0, &state[pixel_index]);
}


__global__ void destroy(Hitable** obj_list, 
                        Hitable** world,
                        Camera** camera, 
                        int obj_cnt) {
    for(int i = 0; i < obj_cnt; i++){
        delete *(obj_list + i);
    }
    delete *world;
    delete *camera;
}


__global__ void render(vec3* colorBuffer,
                       Hitable** world,
                       Camera** camera,
                       hiprandState* state,
                       int nx, 
                       int ny,
                       int samples) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if((x >= nx) || (y >= ny)) return;
    
    int pixel_index = y * nx + x;

    int ns = samples;
    vec3 col(0, 0, 0);
    for(int i = 0; i < ns; i++){
        float u = float(x + rand(&(state[pixel_index]))) / float(nx);
        float v = float(y + rand(&(state[pixel_index]))) / float(ny);
        Ray r = (*camera)->get_ray(u, v, state);
        col += shade(r, world, 0, &(state[pixel_index]));
        // col += shade_nolight(r, world, 0, &(state[pixel_index]));
    }
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);

    colorBuffer[pixel_index] = clip(col);
}


int main() {
    std::freopen("images/image.ppm", "w", stdout);

    int nx = 1024 * RESOLUTION;
    int ny = 512  * RESOLUTION;
    int tx = 16;
    int ty = 16;
    
    int num_pixel = nx * ny;

    // color buffer
    vec3 *colorBuffer;
    checkCudaErrors(hipMallocManaged((void**)& colorBuffer, num_pixel*sizeof(vec3)));

    hiprandState* curand_state;
    checkCudaErrors(hipMallocManaged((void**)& curand_state, num_pixel*sizeof(hiprandState)));

    // build world
    int obj_cnt = 488;
    Hitable** obj_list;
    Hitable** world;
    Camera**  camera;
    checkCudaErrors(hipMallocManaged((void**)& obj_list, obj_cnt*sizeof(Hitable*)));
    checkCudaErrors(hipMallocManaged((void**)& world, sizeof(Hitable*)));
    checkCudaErrors(hipMallocManaged((void**)& camera, sizeof(Camera*)));

    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    random_init <<<blocks, threads>>>(nx, ny, curand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    build_scene <<<1, 1>>>(obj_list, world, camera, curand_state, nx, ny, obj_cnt);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render <<<blocks, threads>>>(colorBuffer, world, camera, curand_state, nx, ny, SAMPLES);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for(int i = ny - 1; i >= 0; i--){
        for(int j = 0; j < nx; j++){
            size_t pixel_index = i * nx + j;
            int ir = int(255.99 * colorBuffer[pixel_index].r());
            int ig = int(255.99 * colorBuffer[pixel_index].g());
            int ib = int(255.99 * colorBuffer[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    destroy<<<1, 1>>>(obj_list, world, camera, obj_cnt);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(obj_list));
    checkCudaErrors(hipFree(camera));
    checkCudaErrors(hipFree(curand_state));
    checkCudaErrors(hipFree(colorBuffer));

    hipDeviceReset();
}