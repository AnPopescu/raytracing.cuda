#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <iostream>

#include "scene.h"


#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

#define RESOLUTION 1
#define SAMPLES 1000


void check_cuda(hipError_t result, 
                char const *const func, 
                const char *const file, 
                int const line){
    if(result){
        std::cerr << "CUDA error = "<< static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const Ray& r, 
                      Hitable **world, 
                      int depth,
                      hiprandState *state) {
    HitRecord rec;
    if((*world)->hit(r, 0.001, MAXFLOAT, rec)){
        Ray scattered;
        vec3 attenuation;
        vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
        if(depth < 30 && rec.mat_ptr->scatter(r, rec, attenuation, scattered, state)){
            return emitted + attenuation * color(scattered, world, depth + 1, state);
        }else{
            return emitted;
        }
    }else{
        return vec3(0, 0, 0);
    }
}


__device__ vec3 color_nolight(const Ray& r, 
                              Hitable **world, 
                              int depth,
                              hiprandState *state) {
    HitRecord rec;
    if((*world)->hit(r, 0.001, MAXFLOAT, rec)){
        Ray scattered;
        vec3 attenuation;
        if(depth < 30 && rec.mat_ptr->scatter(r, rec, attenuation, scattered, state)){
            return attenuation * color_nolight(scattered, world, depth + 1, state);
        }
        else {
            return vec3(0, 0, 0);
        }
    }else{
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5 * (unit_direction.y() + 1.0);
        return (1.0-t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
}


__global__ void build_scene(Hitable** obj_list, 
                            Hitable** world,
                            Camera** camera,
                            hiprandState *state,
                            int nx,
                            int ny,
                            int cnt){
    if(threadIdx.x == 0 && blockIdx.x == 0){
        
        random_scene(obj_list, world, state);
        // simple_light_scene(obj_list, world);
        // cornell_box_bvh_scene(obj_list, world, state);
        // boxlist_bvh_scene(obj_list, world, state);
        // cornell_box_scene(obj_list, world);
        // cornell_smoke_scene(obj_list, world);
        // final_scene(obj_list, world, state);

        // vec3 lookfrom(278, 278, -800);
        // vec3 lookat(278, 278, 0);
        // float dist_to_focus = 10.0;
        // float aperture = 0.0;
        // float vfov = 40.0;

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0;
        float aperture = 0.0;
        float vfov = 20.0;

        *camera = new Camera(lookfrom, 
                             lookat, 
                             vec3(0, 1, 0), 
                             vfov, 
                             float(nx) / float(ny), 
                             aperture, 
                             dist_to_focus,
                             0.0,
                             1.0);
    }
}


__global__ void random_init(int nx, 
                            int ny, 
                            hiprandState *state){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if((x >= nx) || (y >= ny)) return;
    int pixel_index = y * nx + x;
    hiprand_init(0, pixel_index, 0, &state[pixel_index]);
}


__global__ void destroy(Hitable** obj_list, 
                        Hitable** world,
                        Camera** camera, 
                        int obj_cnt) {
    for(int i = 0; i < obj_cnt; i++){
        delete *(obj_list + i);
    }
    delete *world;
    delete *camera;
}


__global__ void render(vec3* fb,
                       Hitable** world,
                       Camera** camera,
                       hiprandState* state,
                       int nx, 
                       int ny){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if((x >= nx) || (y >= ny)) return;
    
    int pixel_index = y * nx + x;   

    int ns = SAMPLES;
    vec3 col(0, 0, 0);
    for(int i = 0; i < ns; i++){
        float u = float(x + rand(&(state[pixel_index]))) / float(nx);
        float v = float(y + rand(&(state[pixel_index]))) / float(ny);
        Ray r = (*camera)->get_ray(u, v, state);
        // col += color(r, world, 0, &(state[pixel_index]));
        col += color_nolight(r, world, 0, &(state[pixel_index]));
    }
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}


int main(){
    std::freopen("images/image.ppm", "w", stdout);

    int nx = 1024 * RESOLUTION;
    int ny = 512  * RESOLUTION;
    int tx = 16;
    int ty = 16;
    
    int num_pixel = nx * ny;

    // final buffer
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void**)& fb, num_pixel*sizeof(vec3)));

    hiprandState* curand_state;
    checkCudaErrors(hipMallocManaged((void**)&curand_state, num_pixel*sizeof(hiprandState)));

    // build world
    int obj_cnt = 488;
    Hitable** obj_list;
    Hitable** world;
    Camera**  camera;
    checkCudaErrors(hipMallocManaged((void**)& obj_list, obj_cnt*sizeof(Hitable*)));
    checkCudaErrors(hipMallocManaged((void**)& world, sizeof(Hitable*)));
    checkCudaErrors(hipMallocManaged((void**)& camera, sizeof(Camera*)));

    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    random_init <<<blocks, threads>>>(nx, ny, curand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    build_scene <<<1, 1>>>(obj_list, world, camera, curand_state, nx, ny, obj_cnt);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render <<<blocks, threads>>>(fb, world, camera, curand_state, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for(int i = ny - 1; i >= 0; i--){
        for(int j = 0; j < nx; j++){
            size_t pixel_index = i * nx + j;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    destroy<<<1, 1>>>(obj_list, world, camera, obj_cnt);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(obj_list));
    checkCudaErrors(hipFree(camera));
    checkCudaErrors(hipFree(curand_state));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}